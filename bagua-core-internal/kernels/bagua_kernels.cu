#include "hip/hip_runtime.h"
#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <nccl.h>
#include <hipcub/hipcub.hpp>
#include "bagua_utils.h"

const float eps = 1e-7;

// Reference: https://github.com/NVIDIA/nccl-tests/blob/2f9bba9f20002e7b7818e7fdeae6e35734260aff/src/common.h#L207
size_t word_size(ncclDataType_t type) {
    switch (type) {
        case ncclChar:
#if NCCL_MAJOR >= 2
            //case ncclInt8:
    case ncclUint8:
#endif
            return 1;
        case ncclHalf:
            //case ncclFloat16:
            return 2;
        case ncclInt:
        case ncclFloat:
#if NCCL_MAJOR >= 2
            //case ncclInt32:
    case ncclUint32:
    //case ncclFloat32:
#endif
            return 4;
        case ncclInt64:
        case ncclUint64:
        case ncclDouble:
            //case ncclFloat64:
            return 8;
        default:
            return 0;
    }
}

// Reference: https://github.com/NVIDIA/TensorRT/blob/master/plugin/common/common.cuh#L67
__device__ inline half __hdiv_with_fallback(const half a, const half b) {
#if __CUDA_ARCH__ >= 530
    return __hdiv(a, b);
#else
    float out;
    out = __half2float(a) / __half2float(b);
    return __float2half_rn(out);
#endif
}

__device__ inline half __havg_with_fallback(const half a, const half b) {
#if __CUDA_ARCH__ >= 530
    return __hadd(a, b) / __float2half(2.0);
#else
    float out;
    out = (__half2float(a) + __half2float(b)) / 2.0;
    return __float2half_rn(out);
#endif
}

__device__ inline half __hsub_with_fallback(const half a, const half b) {
#if __CUDA_ARCH__ >= 530
    return __hsub(a, b);
#else
    float out;
    out = __half2float(a) - __half2float(b);
    return __float2half_rn(out);
#endif
}

__device__ inline half __hadd_with_fallback(const half a, const half b) {
#if __CUDA_ARCH__ >= 530
    return __hadd(a, b);
#else
    float out;
    out = __half2float(a) + __half2float(b);
    return __float2half_rn(out);
#endif
}

__device__ inline half __haddmul_with_fallback(const half a, const half b, const half factor) {
#if __CUDA_ARCH__ >= 530
    return __hadd(a, __hmul(b, factor));
#else
    float out;
    out = __half2float(a) + __half2float(b) * __half2float(factor);
    return __float2half_rn(out);
#endif
}

// Reference: https://github.com/dmlc/cub/blob/master/cub/thread/thread_operators.cuh
struct Sum {
    /// Boolean sum operator, returns <tt>a + b</tt>
    template<typename T>
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

struct Max
{
    template <typename T>
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return ((b > a) ? b : a);
    }

};

template <>
__device__ __forceinline__ half Max::operator()<half>(const half &a, const half &b) const {
#if __CUDA_ARCH__ >= 530
	return __hgt(b, a) ? b: a;
#else
	return (__half2float(b) > __half2float(a) ? b : a);
#endif
}


struct Min
{
    template <typename T>
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return ((b < a) ? b : a);
    }

};

template <>
__device__ __forceinline__ half Min::operator()<half>(const half &a, const half &b) const {
#if __CUDA_ARCH__ >= 530
    return __hlt(b, a) ? b: a;
#else
    return (__half2float(b) < __half2float(a) ? b : a);
#endif
}


template<typename T>
__device__ inline float __accum_to_float(float a, T b) {
    return a + b;
}

template<>
__device__ inline float __accum_to_float<half>(float a, half b) {
    return a + __half2float(b);
}

template<typename T, bool average>
__device__ inline T __from_float(float a, int n, T placeholder) {
   if (average) {
       return a / n;
   } else {
       return a;
   }
}

template<>
__device__ inline half __from_float<half, true>(float a, int n, half placeholder) {
   return  __float2half(a / n);
}

template<>
__device__ inline half __from_float<half, false>(float a, int n, half placeholder) {
   return  __float2half(a);
}

template<typename ReductionOpT, unsigned int block_dim_y>
__device__ void
block_y_reduce(float sdata[][block_dim_y], unsigned int tidx, unsigned int tidy, ReductionOpT reduction_op) {
    if (block_dim_y >= 32) {
        if (tidy < 16) { sdata[tidx][tidy] = reduction_op(sdata[tidx][tidy], sdata[tidx][tidy + 16]); }
        __syncthreads();
    }
    if (block_dim_y >= 16) {
        if (tidy < 8) { sdata[tidx][tidy] = reduction_op(sdata[tidx][tidy], sdata[tidx][tidy + 8]); }
        __syncthreads();
    }
    if (block_dim_y >= 8) {
        if (tidy < 4) { sdata[tidx][tidy] = reduction_op(sdata[tidx][tidy], sdata[tidx][tidy + 4]); }
        __syncthreads();
    }
    if (block_dim_y >= 4) {
        if (tidy < 2) { sdata[tidx][tidy] = reduction_op(sdata[tidx][tidy], sdata[tidx][tidy + 2]); }
        __syncthreads();
    }
    if (block_dim_y >= 2) {
        if (tidy < 1) { sdata[tidx][tidy] = reduction_op(sdata[tidx][tidy], sdata[tidx][tidy + 1]); }
        __syncthreads();
    }
}

__global__ void average_inplace_f32(float *x, float *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = (x[i] + y[i]) / 2.0;
    }
}

__global__ void average_inplace_f16(__half *x, __half *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = __havg_with_fallback(x[i], y[i]);
    }
}

__global__ void substract_inplace_f32(float *x, float *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] -= y[i];
    }
}

__global__ void substract_inplace_f16(__half *x, __half *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = __hsub_with_fallback(x[i], y[i]);
    }
}

__global__ void add_inplace_f32(float *x, float *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] += y[i];
    }
}

__global__ void add_inplace_f16(__half *x, __half *y, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = __hadd_with_fallback(x[i], y[i]);
    }
}

__global__ void addmul_inplace_f32(float *x, float *y, int N, float factor) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] += y[i] * factor;
    }
}

__global__ void addmul_inplace_f16(__half *x, __half *y, int N, __half factor) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = __haddmul_with_fallback(x[i], y[i], factor);
    }
}

__global__ void divide_inplace_f32(float *x, float D_, int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = x[i] / D_;
    }
}

__global__ void divide_inplace_f16(__half *x, float D_, int N) {
    __half D__ = __float2half(D_);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        x[i] = __hdiv_with_fallback(x[i], D__);
    }
}

__global__ void async_model_average(float *tensor, const float *reduced_tensor_copy, 
		const float *tensor_copy, const float nranks, const int N) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
	
	tensor[i] += reduced_tensor_copy[i] / nranks - tensor_copy[i];
        /*if (tensor[i] != tensor[i]) {
            printf("nan encountered!");
        }*/
//        atomicAdd(&tensor[i], reduced_tensor_copy[i] / nranks - tensor_copy[i]);
    }
}

template<typename T>
size_t array_min_max_size(
        const T *input_array,
        int num_items,
        T *output_array,
        hipStream_t stream) {

     void *dev_buffer = NULL;
     size_t dev_buffer_bytes = 0;

    CUDACHECK(hipcub::DeviceReduce::Min(
                dev_buffer,
                dev_buffer_bytes,
                input_array,
                output_array,
                num_items,
                stream));

    return dev_buffer_bytes;
}

template<>
size_t array_min_max_size<half>(
        const half *input_array,
        int num_items,
        half *output_array,
        hipStream_t stream) {

    void *dev_buffer = NULL;
    size_t dev_buffer_bytes = 0;

    hipcub::DeviceReduce::Reduce(
            dev_buffer,
            dev_buffer_bytes,
            input_array,
            output_array,
            num_items,
            Min(),
            __float2half(65504),  // FIXME
            stream);

    return dev_buffer_bytes;
}

template<typename T>
void array_min_max(
        const T *input_array,
        int num_items,
        void *dev_buffer,
        size_t dev_buffer_bytes,
        T *output_array,
        hipStream_t stream) {

    CUDACHECK(hipcub::DeviceReduce::Min(
                dev_buffer,
                dev_buffer_bytes,
                input_array,
                output_array,
                num_items,
                stream));
    
    CUDACHECK(hipcub::DeviceReduce::Max(
            dev_buffer,
            dev_buffer_bytes,
            input_array,
            output_array + 1,
            num_items,
            stream));


//    float *val = (float *) malloc(sizeof(float));
//    hipMemcpy(val, output_max, sizeof(float), hipMemcpyDeviceToHost);
//    std::cout << "max " << *val << std::endl;
}

template<>
void array_min_max<half>(
        const half *input_array,
        int num_items,
        void *dev_buffer,
        size_t dev_buffer_bytes,
        half *output_array,
        hipStream_t stream) {

    hipcub::DeviceReduce::Reduce(
            dev_buffer,
            dev_buffer_bytes,
            input_array,
            output_array,
            num_items,
            Min(),
            __float2half(65504),  // FIXME
            stream);

    hipcub::DeviceReduce::Reduce(
            dev_buffer,
            dev_buffer_bytes,
            input_array,
            output_array + 1,
            num_items,
            Max(),
            __float2half(-65504),  // FIXME
            stream);
}

template<unsigned int block_dim_x, unsigned int block_dim_y, typename T, bool average>
__global__ void reduce_chunk_inplace(T *input, int chunk_size, int num_chunks, int target_chunk) {

    __shared__ float sdata[block_dim_x][block_dim_y];

    unsigned int tidx = threadIdx.x;
    unsigned int tidy = threadIdx.y;

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // load to shared memory
    float sum = 0.0;
    for (int i = idy; i < num_chunks && idx < chunk_size; i += blockDim.y) {
        sum = __accum_to_float(sum, input[chunk_size * i + idx]);
    }

    sdata[tidx][tidy] = sum;
    __syncthreads();

    block_y_reduce<Sum, block_dim_y>(sdata, tidx, tidy, Sum());

    // write to global memory
    T *output = input + target_chunk * chunk_size;
    if (tidy == 0 && idx < chunk_size) {
        output[idx] = __from_float<T, average>(sdata[tidx][tidy], num_chunks, output[idx]);
    }
}

template<typename T>
__device__ inline uint8_t __minmax_uint8_compress(T f, float scale, float lower_bound, float upper_bound) {
    float level = f * scale;
    level = min(level, upper_bound);
    return level - lower_bound;

}

template<>
__device__ inline uint8_t __minmax_uint8_compress<float>(float f, float scale, float lower_bound, float upper_bound) {
    float level = rintf(f * scale);
    level = min(level, upper_bound);
    return level - lower_bound;
}

template<>
__device__ inline uint8_t __minmax_uint8_compress<half>(half f, float scale, float lower_bound, float upper_bound) {
    float level = rintf(__half2float(f) * scale);
    level = min(level, upper_bound);
    return level - lower_bound;
}

template<typename T>
__device__ inline T __minmax_uint8_decompress(uint8_t i, float scale, float lower_bound, float upper_bound, T placeholder) {
    return (i + lower_bound) / scale;
}

template<>
__device__ inline half __minmax_uint8_decompress<half>(uint8_t i, float scale, float lower_bound, float upper_bound, half placeholder) {
    return __float2half((i + lower_bound) / scale);
}

template<typename T>
__device__ inline float __load_as_float(T * array) {
    return array[0];
}

template<>
__device__ inline float __load_as_float<half>(half * array) {
    return __half2float(array[0]);
}

template<typename T>
__device__ inline void __store_float(T * array, float data) {
    array[0] = data;
}

template<>
__device__ inline void __store_float<half>(half * array, float data) {
    array[0] = __float2half(data);
}


template<typename T>
__global__ void
compress_float_to_uint8(T *input, int chunk_size, int chunk_offset, int num_chunks, uint8_t *output,
                      size_t output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float min_ = __load_as_float(reinterpret_cast<T *>(output + idy * chunk_offset));
    float max_ = __load_as_float(reinterpret_cast<T *>(output + idy * chunk_offset + sizeof(T)));

    float scale = 255.0 / (max_ - min_ + eps);
    float upper_bound = rintf(max_ * scale);
    float lower_bound = upper_bound - 255.0;
    for (int i = idx; i < chunk_size; i += blockDim.x * gridDim.x) {
        int k = idy * chunk_size + i;
        int o = idy * chunk_offset + 32 + i;
        output[o] = __minmax_uint8_compress(input[k], scale, lower_bound, upper_bound);
    }

    if (idx == 0) {
        // write max min to output buffer
        __store_float(reinterpret_cast<T *>(output + idy * chunk_offset), min_);
        __store_float(reinterpret_cast<T *>(output + idy * chunk_offset + sizeof(T)), max_);
    }
}

template<typename T>
__global__ void
decompress_uint8_to_float(uint8_t *input, size_t input_size, int chunk_size, int chunk_offset, int num_chunks,
                          T *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const float min_ = __load_as_float(reinterpret_cast<T *>(input + idy * chunk_offset));
    const float max_ = __load_as_float(reinterpret_cast<T *>(input + idy * chunk_offset + sizeof(T)));

    float scale = 255.0 / (max_ - min_ + eps);
    float upper_bound = rintf(max_ * scale);
    float lower_bound = upper_bound - 255.0;

    for (int i = idx; i < chunk_size; i += blockDim.x * gridDim.x) {
        int k = idy * chunk_size + i;
        int o = idy * chunk_offset + 32 + i;
        output[k] = __minmax_uint8_decompress(input[o], scale, lower_bound, upper_bound, output[k]);
    }
}

template<typename T, bool average>
void reduce_chunk_inplace_host(T *input, int chunk_size, int num_chunks, int target_chunk, hipStream_t stream) {
    if (num_chunks <= 4) {
        dim3 num_blocks(DIVUP(chunk_size, 512), 1);
        dim3 num_threads(512, 2);
        reduce_chunk_inplace<512, 2, T, average><<<num_blocks, num_threads, 0, stream>>>(input, chunk_size, num_chunks,
                                                                               target_chunk);
    } else if (num_chunks <= 8) {
        dim3 num_blocks(DIVUP(chunk_size, 256), 1);
        dim3 num_threads(256, 4);
        reduce_chunk_inplace<256, 4, T, average><<<num_blocks, num_threads, 0, stream>>>(input, chunk_size, num_chunks,
                                                                               target_chunk);
    } else if (num_chunks <= 16) {
        dim3 num_blocks(DIVUP(chunk_size, 128), 1);
        dim3 num_threads(128, 8);
        reduce_chunk_inplace<128, 8, T, average><<<num_blocks, num_threads, 0, stream>>>(input, chunk_size, num_chunks,
                                                                               target_chunk);
    } else if (num_chunks <= 32) {
        dim3 num_blocks(DIVUP(chunk_size, 64), 1);
        dim3 num_threads(64, 16);
        reduce_chunk_inplace<64, 16, T, average><<<num_blocks, num_threads, 0, stream>>>(input, chunk_size, num_chunks,
                                                                               target_chunk);
    } else {
        dim3 num_blocks(DIVUP(chunk_size, 32), 1);
        dim3 num_threads(32, 32);
        reduce_chunk_inplace<32, 32, T, average><<<num_blocks, num_threads, 0, stream>>>(input, chunk_size, num_chunks,
                                                                               target_chunk);
    }
    CUDACHECK(hipGetLastError());
}

template<typename T>
void compress_float_to_uint8_host(T *input, int input_num_element, int chunk_size, int num_chunks, uint8_t *output,
                                size_t output_size, void *dev_buffer, size_t dev_size, int target_chunk,
                                hipStream_t stream) {
    int chunk_offset = output_size / num_chunks;
    int remaining_elem = input_num_element;
    for (int i = 0; i < num_chunks; i++) {
        if ((target_chunk == -1) || (i == target_chunk)) {
            array_min_max(input + i * chunk_size, std::min(remaining_elem, chunk_size), dev_buffer, dev_size,
                          reinterpret_cast<T *>(output + i * chunk_offset), stream);
        }
        remaining_elem -= chunk_size;
    }

    if (target_chunk == -1) {
        dim3 num_blocks(DIVUP(chunk_size, 1024), num_chunks);
        compress_float_to_uint8<<<num_blocks, 1024, 0, stream>>>(input, chunk_size, chunk_offset, num_chunks, output,
                                                               output_size);
    } else {
        dim3 num_blocks(DIVUP(chunk_size, 1024), 1);
        T *chunk_input = input + target_chunk * chunk_size;
        uint8_t *chunk_output = output + target_chunk * chunk_offset;

        compress_float_to_uint8<<<num_blocks, 1024, 0, stream>>>(chunk_input, chunk_size, chunk_offset, 1, chunk_output,
                                                               chunk_offset);
    }
    CUDACHECK(hipGetLastError());
}

template<typename T>
void decompress_uint8_to_float_host(uint8_t *input, size_t input_size, int chunk_size, int num_chunks, T *output,
                                   hipStream_t stream) {

    int chunk_offset = input_size / num_chunks;
    dim3 num_blocks(DIVUP(chunk_size, 1024), num_chunks);
    decompress_uint8_to_float<<<num_blocks, 1024, 0, stream>>>(input, input_size,
                                                             chunk_size, chunk_offset, num_chunks, output);
    CUDACHECK(hipGetLastError());
}

extern "C" {
void divide_inplace_f32_host(float *x, float D_, int N, hipStream_t stream) {
    divide_inplace_f32<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, D_, N);
    CUDACHECK(hipGetLastError());
}

void divide_inplace_f16_host(__half *x, float D_, int N, hipStream_t stream) {
    divide_inplace_f16<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, D_, N);
    CUDACHECK(hipGetLastError());
}

void add_inplace_f32_host(float *x, float *y, int N, hipStream_t stream) {
    add_inplace_f32<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void add_inplace_f16_host(__half *x, __half *y, int N, hipStream_t stream) {
    add_inplace_f16<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void addmul_inplace_f32_host(float *x, float *y, int N, const float factor, hipStream_t stream) {
    addmul_inplace_f32<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N, factor);
    CUDACHECK(hipGetLastError());
}

void addmul_inplace_f16_host(__half *x, __half *y, int N, const float factor, hipStream_t stream) {
    addmul_inplace_f16<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N, __float2half(factor));
    CUDACHECK(hipGetLastError());
}

void substract_inplace_f32_host(float *x, float *y, int N, hipStream_t stream) {
    substract_inplace_f32<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void substract_inplace_f16_host(__half *x, __half *y, int N, hipStream_t stream) {
    substract_inplace_f16<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void average_inplace_f32_host(float *x, float *y, int N, hipStream_t stream) {
    average_inplace_f32<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void average_inplace_f16_host(__half *x, __half *y, int N, hipStream_t stream) {
    average_inplace_f16<<<DIVUP(N, 1024), 1024, 0, stream>>>(x, y, N);
    CUDACHECK(hipGetLastError());
}

void async_model_average_host(float *tensor, const float *reduced_tensor_copy, 
		const float *tensor_copy, const float nranks, const int N, hipStream_t stream) {
    async_model_average<<<DIVUP(N, 1024), 1024, 0, stream>>>(tensor, reduced_tensor_copy, tensor_copy, nranks, N);
    CUDACHECK(hipGetLastError());
}

//// decentralize, recvbuf should get the average of sendbuf and peer's sendbuf
//ncclResult_t ncclPeerAverage(void *sendbuf, void *recvbuf, size_t sendcount,
//                             int peer_rank, ncclDataType_t datatype, ncclComm_t comm, hipStream_t stream) {
//    NCCLCHECK(ncclGroupStart());
//    NCCLCHECK(ncclSend(sendbuf, sendcount, datatype, peer_rank, comm, stream));
//    NCCLCHECK(ncclRecv(recvbuf, sendcount, datatype, peer_rank, comm, stream));
//    NCCLCHECK(ncclGroupEnd());
//
//    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < sendcount; i += blockDim.x * gridDim.x) {
//        // FIXME: this is not always float
////        float f = sendbuf[i];
////        sendbuf[i] = (sendbuf[i] + recvbuf[i]) / 2;
//    }
//    return ncclSuccess;
//}

void reduce_mean_f32_inplace_host(float *input, int chunk_size, int num_chunks, int target_chunk, hipStream_t stream) {
    reduce_chunk_inplace_host<float, true>(input, chunk_size, num_chunks, target_chunk, stream);
}

void reduce_mean_f16_inplace_host(half *input, int chunk_size, int num_chunks, int target_chunk, hipStream_t stream) {
    reduce_chunk_inplace_host<half, true>(input, chunk_size, num_chunks, target_chunk, stream);
}

void reduce_sum_f32_inplace_host(float *input, int chunk_size, int num_chunks, int target_chunk, hipStream_t stream) {
    reduce_chunk_inplace_host<float, false>(input, chunk_size, num_chunks, target_chunk, stream);
}

void reduce_sum_f16_inplace_host(half *input, int chunk_size, int num_chunks, int target_chunk, hipStream_t stream) {
    reduce_chunk_inplace_host<half, false>(input, chunk_size, num_chunks, target_chunk, stream);
}
void compress_f32_to_uint8_host(float *input, int input_num_element, int chunk_size, int num_chunks, uint8_t *output,
                                size_t output_size, void *dev_buffer, size_t dev_size, int target_chunk,
                                hipStream_t stream) {
    compress_float_to_uint8_host(input, input_num_element, chunk_size, num_chunks, output, output_size, dev_buffer, dev_size, target_chunk, stream);
}

void decompress_uint8_to_f32_host(uint8_t *input, size_t input_size, int chunk_size, int num_chunks, float *output,
                                  hipStream_t stream) {
    decompress_uint8_to_float_host(input, input_size, chunk_size, num_chunks, output, stream);
}

void compress_f16_to_uint8_host(half *input, int input_num_element, int chunk_size, int num_chunks, uint8_t *output,
                                size_t output_size, void *dev_buffer, size_t dev_size, int target_chunk,
                                hipStream_t stream) {
    compress_float_to_uint8_host(input, input_num_element, chunk_size, num_chunks, output, output_size, dev_buffer, dev_size, target_chunk, stream);
}

void decompress_uint8_to_f16_host(uint8_t *input, size_t input_size, int chunk_size, int num_chunks, half *output, 
		                  hipStream_t stream) {
    decompress_uint8_to_float_host(input, input_size, chunk_size, num_chunks, output, stream);
}

size_t array_min_max_size_f32_host(float *input, int input_num_element, float *output, hipStream_t stream) {
    return array_min_max_size(input, input_num_element, output, stream);
}

size_t array_min_max_size_f16_host(half *input, int input_num_element, half *output, hipStream_t stream) {
    return array_min_max_size(input, input_num_element, output, stream);
}

}

ncclResult_t ncclAllToAll(void *sendbuf,
                          void *recvbuf,
                          size_t count,
                          ncclDataType_t datatype,
                          ncclComm_t comm,
                          int nranks,
                          int rank,
                          hipStream_t stream) {
    if (sendbuf == recvbuf) {
        return ncclInvalidUsage;
    }

    // awkward workaround for nvcc bug
    intptr_t sendbuff = reinterpret_cast<intptr_t>(sendbuf);
    intptr_t recvbuff = reinterpret_cast<intptr_t>(recvbuf);
    NCCLCHECK(ncclGroupStart());
    for (int r = 0; r < nranks; ++r) {
        intptr_t r_sendbuf = sendbuff + r * count * word_size(datatype);
        intptr_t r_recvbuf = recvbuff + r * count * word_size(datatype);
        if (r != rank) {
            // awkward workaround for nvcc bug
            int peer = (int) r;
            NCCLCHECK(ncclSend(reinterpret_cast<const void *>(r_sendbuf), count, datatype, peer, comm, stream));
            NCCLCHECK(ncclRecv(reinterpret_cast<void *>(r_recvbuf), count, datatype, peer, comm, stream));
        } else {
            CUDACHECK(hipMemcpyAsync(reinterpret_cast<void *>(r_recvbuf), reinterpret_cast<const void *>(r_sendbuf),
                            count * word_size(datatype), hipMemcpyDeviceToDevice, stream));
        }
    }
    NCCLCHECK(ncclGroupEnd());
    return ncclSuccess;
}
